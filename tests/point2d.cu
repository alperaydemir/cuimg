#include <cassert>
#include <cuimg/point2d.h>

using namespace cuimg;

int main()
{
  point2d<int> p(300, 200);

  assert(p.row() == 300);
  assert(p.col() == 200);

  point2d<int> o(3, 2);

  p = o;
  assert(p.row() == 3);
  assert(p.col() == 2);

}
