
#include <cassert>
#include <cuimg/point2d.h>
#include <cuimg/obox2d.h>

using namespace cuimg;

int main()
{
  obox2d<point2d<int> > b(300, 200);

  assert(b.nrows() == 300);
  assert(b.ncols() == 200);

  assert(b.has(point2d<int>(0, 0)));
  assert(!b.has(point2d<int>(-1, 0)));
  assert(!b.has(point2d<int>(0, -1)));
  assert(b.has(point2d<int>(299, 199)));
  assert(!b.has(point2d<int>(300, 199)));
  assert(!b.has(point2d<int>(299, 200)));
}
