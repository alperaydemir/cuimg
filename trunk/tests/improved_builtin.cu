#include "hip/hip_runtime.h"
#include <cassert>
#include <cuimg/improved_builtin.h>

using namespace cuimg;

int main()
{
  {
    i_uchar4 t(make_i_float4(1,1,2,3));
    assert(t == i_uchar4(1, 1, 2, 3));
    assert(t != i_uchar4(2, 1, 2, 3));

    i_uchar4 u(make_uchar4(1,2,3,4));
    assert(u == i_uchar4(1, 2, 3, 4));
    assert(u == i_float4(1, 2, 3, 4));

    t = i_uchar4(1,2,3,4);
    assert(t == i_uchar4(1, 2, 3, 4));

    t += u;
    assert(t == i_uchar4(2, 4, 6, 8));
    t -= u;
    assert(t == i_uchar4(1, 2, 3, 4));
    t *= 2;
    assert(t == i_uchar4(2, 4, 6, 8));
    t /= 2;
    assert(t == i_uchar4(1, 2, 3, 4));

    t = u + u;
    assert(t == i_uchar4(2, 4, 6, 8));
    t = t - u;
    assert(t == i_uchar4(1, 2, 3, 4));
    t = t * 2;
    assert(t == i_uchar4(2, 4, 6, 8));
    t = t / 2;
    assert(t == i_uchar4(1, 2, 3, 4));
  }

  {
    i_uchar4 t(0,0,0,0);
    i_uchar4 u(0,0,0,0);
    i_char1 u1(41);
    i_char2 u2(41, 41);
    i_char3 u3(41, 41, 41);
    i_float4 u4(41, 41, 4, 41);

    float x = u4[2];
    assert(x == 4);

    i_float4 z = t + u4;

    i_uchar4 uchar_test = z;
    z = u4;
    z = t + u4;

    assert(z == i_float4(41, 41, 4, 41));
    z = t - u4;
    assert(z == i_float4(-41, -41, -4, -41));
    z = i_float4(2,4,6,8);
    z = z / 2;
    assert(z == i_float4(1,2,3,4));
    z = z * 2;
    assert(z == i_float4(2,4,6,8));

    assert(u4 == make_i_float4(41, 41, 4, 41));
    assert(u4 == make_i_char4(41, 41, 4, 41));
    assert(u4 != make_i_float4(41, 41, 5, 41));
    assert(u4 != make_i_char4(41, 41, 5, 41));

    assert(u4 != make_i_char4(41, 5, 41, 2));

    u.w = 4;
    t = u;
    assert(t.w == 4);
    t += make_i_char4(0,0,0,0);
    t -= make_i_char4(0,0,0,0);
    t *= 42;
    t /= 42;
  }
}
