#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <cmath>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>


#include <hip/hip_runtime.h>

#include <cuimg/improved_builtin.h>
#include <cuimg/builtin_math.h>
#include <cuimg/image2d.h>
#include <cuimg/copy.h>
#include <cuimg/kernel_image2d.h>
#include <cuimg/host_image2d.h>
#include <cuimg/neighb2d_data.h>
#include <cuimg/neighb_iterator2d.h>
#include <cuimg/static_neighb2d.h>
#include <cuimg/texture.h>
#include <cuimg/convolve.h>

using namespace cuimg;

#define IMG_SIZE 3072
#define KERNEL_SIZE 15
#define VTYPE_CUDA float1
#define VTYPE i_float1
#define CONCAT(A, B, C) A ## B ## C
#define XCONCAT(A, B, C) CONCAT(A, B, C)
#define DPOINTS_CPU XCONCAT(c, KERNEL_SIZE ,_rows_cpu)
#define DPOINTS_GPU XCONCAT(c, KERNEL_SIZE ,_rows)
#define ITERATIONS_GPU 100

template <typename T>
struct tex2d;
REGISTER_TEXTURE2D_PROXY(tex2d);


texture<float, 1, hipReadModeElementType> kernel_weights;
texture<int2, 1, hipReadModeElementType> kernel_dpoints;

// convolutions 1d:
//   weights: texture, dpoints: texture or static array?

__constant__ const int c3_rows[3][2] = {{-1, 0}, {0, 0}, {1, 0}};
         const int c3_rows_cpu[3][2] = {{-1, 0}, {0, 0}, {1, 0}};

//__constant__ const int c9_rows[9][2] = {{0, -4}, {0, -3}, {0, -2}, {0, -1}, {0, 0}, {0, 1}, {0, 2}, {0, 3}, {0, 4}};
//         const int c9_rows_cpu[9][2] = {{0, -4}, {0, -3}, {0, -2}, {0, -1}, {0, 0}, {0, 1}, {0, 2}, {0, 3}, {0, 4}};

__constant__ const int c9_rows[9][2] = {{-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}};
         const int c9_rows_cpu[9][2] = {{-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}};

__constant__ const int c15_rows[15][2] = {{-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}};
         const int c15_rows_cpu[15][2] = {{-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}};

__constant__ const int c31_rows[31][2] = {{-15, 0}, {-14, 0}, {-13, 0}, {-12, 0}, {-10, 0}, {-9, 0}, {-8, 0}, {-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0},{-1, 0},
                                  {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}, {8, 0}, {9, 0}, {10, 0}, {11, 0}, {12, 0}, {13, 0}, {14, 0}, {15, 0}};
         const int c31_rows_cpu[31][2] = {{-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}};


class weigthed_window
{
public:

  weigthed_window(point2d<int>* dpoints,
                  float* weights,
                  unsigned size)
                  : dpoints_(dpoints),
                    weights_(weights),
                    size_(size)
  {
  }

  point2d<int> dpoints(int i)const { return dpoints_[i]; }
  float weights(int i) const { return weights_[i]; }
  unsigned size() const { return size_; }

private:
  point2d<int>* dpoints_;
  float* weights_;
  unsigned size_;
};

template <typename T>
__global__ void convolve_rows_static_global(kernel_image2d<T> out, float* weights)
{
  //int idr = blockIdx.x * blockDim.x + threadIdx.x;
  //int idc = blockIdx.y * blockDim.y + threadIdx.y;
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  bt_change_vtype(T, type_mult(bt_vtype(T), float)) r  = zero();
  neighb_iterator2d<static_neighb2d<KERNEL_SIZE> > n(p, static_neighb2d<KERNEL_SIZE>(DPOINTS_GPU));
  for_all(n) if (out.has(n))
    r += T(tex2D(tex2d<VTYPE_CUDA>::tex(), n->col(), n->row())) * weights[n.i()];
  out(p) = r;
}

template <typename T>
__global__ void convolve_rows_static_texture(kernel_image2d<T> out)
{
  //int idr = blockIdx.x * blockDim.x + threadIdx.x;
  //int idc = blockIdx.y * blockDim.y + threadIdx.y;
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  bt_change_vtype(T, type_mult(bt_vtype(T), float)) r  = zero();
  neighb_iterator2d<static_neighb2d<KERNEL_SIZE> > n(p, static_neighb2d<KERNEL_SIZE>(DPOINTS_GPU));
  for_all(n) if (out.has(n))
  {
    float w = tex1Dfetch(kernel_weights, n.i());
    r += T(tex2D(tex2d<VTYPE_CUDA>::tex(), n->col(), n->row())) * w;
  }
  out(p) = r;
}

template <typename T>
__global__ void convolve_rows_texture_texture(kernel_image2d<T> out, unsigned kernelsize)
{
  //int idr = blockIdx.x * blockDim.x + threadIdx.x;
  //int idc = blockIdx.y * blockDim.y + threadIdx.y;
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  bt_change_vtype(T, type_mult(bt_vtype(T), float)) r  = zero();
  for(int i = 0; i < kernelsize; i++)
  {
    float w = tex1Dfetch(kernel_weights, i);
    point2d<int> n = i_int2(tex1Dfetch(kernel_dpoints, i)) + p;
    if (out.has(n))
      r += T(tex2D(tex2d<VTYPE_CUDA>::tex(), n.col(), n.row())) * w;
  }
  out(p) = r;
}

template <typename T>
__global__ void convolve_rows_loop_texture(kernel_image2d<T> out)
{
  //int idr = blockIdx.x * blockDim.x + threadIdx.x;
  //int idc = blockIdx.y * blockDim.y + threadIdx.y;
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  bt_change_vtype(T, type_mult(bt_vtype(T), float)) r  = zero();
  for(int i = 0; i < KERNEL_SIZE; i++)
  {
    float w = tex1Dfetch(kernel_weights, i);
    point2d<int> n = i_int2(i - KERNEL_SIZE/2, 0) + p;
    if (out.has(n))
      r += T(tex2D(tex2d<VTYPE_CUDA>::tex(), n.col(), n.row())) * w;
  }
  out(p) = r;
}

template <int i>
struct convolve_rows_unrolled_texture_loop
{
  template <typename U, typename T, unsigned N>
  static __device__ void run(const i_int2& p, kernel_image2d<U>& out, improved_builtin<T, N>& r)
  {
    float w = tex1Dfetch(kernel_weights, i);
    point2d<int> n = i_int2(i - KERNEL_SIZE/2, 0) + p;
    //if (out.has(n))
      r += U(tex2D(tex2d<VTYPE_CUDA>::tex(), n.col(), n.row())) * w;
//    int col = p.y + i - KERNEL_SIZE/2;
//    if (col >= 0 && col < out.ncols())
//      r += U(tex2D(tex2d<VTYPE_CUDA>::tex(), col, p.x)) * w;
  }
};


template <int i, int E>
struct convolve_rows_unrolled_texture_loop_inline
{
  template <typename U>
  static __device__ U run(const i_int2& p, kernel_image2d<U>& out)
  {
      return U(tex2D(tex2d<VTYPE_CUDA>::tex(), p.y, p.x + i - KERNEL_SIZE/2)) * 0.132
        + convolve_rows_unrolled_texture_loop_inline<i + 1, E>::run(p, out);
  }
};

template <int E>
struct convolve_rows_unrolled_texture_loop_inline<E, E>
{
  template <typename U>
  static __device__ U run(const i_int2& p, kernel_image2d<U>& out)
  {
      return U(tex2D(tex2d<VTYPE_CUDA>::tex(), p.y, p.x + E - KERNEL_SIZE/2));
  }
};


template <typename T>
__global__ void convolve_rows_unrolled_texture_special(kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  bt_change_vtype(T, type_mult(bt_vtype(T), float)) r  = zero();
/*
  convolve_rows_unrolled_texture_loop<31>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<0>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<27>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<1>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<25>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<3>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<23>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<5>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<21>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<7>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<19>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<9>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<17>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<11>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<15>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<16>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<13>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<18>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<14>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<20>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<12>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<22>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<10>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<24>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<8>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<26>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<6>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<28>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<4>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<30>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<2>::run(p, out, r);
  */


  convolve_rows_unrolled_texture_loop<0>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<2>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<1>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<4>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<3>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<6>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<5>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<8>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<7>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<10>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<9>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<12>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<11>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<14>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<13>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<16>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<15>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<18>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<17>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<20>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<19>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<22>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<21>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<24>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<23>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<26>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<25>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<28>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<27>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<30>::run(p, out, r);
  convolve_rows_unrolled_texture_loop<31>::run(p, out, r);

  //meta::special_loop<convolve_rows_unrolled_texture_loop, 0, KERNEL_SIZE - 1>::iter(p, out, r);
  out(p) = r;
}

template <typename T>
__global__ void convolve_rows_unrolled_texture(kernel_image2d<T> out)
{
  //int idr = blockIdx.x * blockDim.x + threadIdx.x;
  //int idc = blockIdx.y * blockDim.y + threadIdx.y;
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  bt_change_vtype(T, type_mult(bt_vtype(T), float)) r  = zero();
//  meta::loop<convolve_rows_unrolled_texture_loop, 0, KERNEL_SIZE - 1>::iter(p, out, r);
  out(p) = convolve_rows_unrolled_texture_loop_inline<0, KERNEL_SIZE - 1>::run(p, out);
}

template <typename T>
void convolve_cpu(host_image2d<T>& in, host_image2d<T>& out, float* weights)
{
  for (unsigned r = 0; r < in.nrows(); r++)
    for (unsigned c = 0; c < in.ncols(); c++)
    {
      point2d<int> p(r, c);
      bt_change_vtype(T, type_mult(bt_vtype(T), float)) res  = zero();
      for(unsigned i = 0; i < KERNEL_SIZE; i++)
      {
        point2d<int> n = i_int2(DPOINTS_CPU[i][0], DPOINTS_CPU[i][1]) + i_int2(p);
        if (in.has(n))
          res += in(n) * weights[i];
      }
      out(p) = res;
    }
}

template <typename T>
void reset(host_image2d<T>& in)
{
  memset(in.data(), 0, in.domain().nrows() * in.domain().ncols() * sizeof(T));
}

template <typename T>
void reset(device_image2d<T>& in)
{
  hipMemset(in.data(), 0, in.domain().nrows() * in.pitch());
}

template <typename T>
float diff(host_image2d<T>& a, host_image2d<T>& b)
{
  float res = 0;
  for(unsigned i = 0; i < a.nrows(); i++)
    for(unsigned j = 0; j < a.ncols(); j++)
      res += norml2(a(i, j) - b(i, j));
  return res;
}

template <typename T>
void stats_diff(host_image2d<T>& a, host_image2d<T>& b)
{
  float max = 0;
  float sum = 0;
  for(unsigned i = 0; i < a.nrows(); i++)
    for(unsigned j = 0; j < a.ncols(); j++)
    {
      float d = norml2(a(i, j) - b(i, j));
      sum += d;
      if (d > max) max = d;
    }

  std::cout << "max diff per pixel: " << max << std::endl;
  std::cout << "sum: " << sum << std::endl;
  std::cout << "mean: " << sum / (a.nrows() * a.ncols()) << std::endl;
}

template <typename T>
void print(const host_image2d<T>& a)
{
  if (a.nrows() * a.ncols() > 20)
    return;

  std::cout << "-----------------------" << std::endl;
  for(unsigned i = 0; i < a.nrows(); i++)
  {
    for(unsigned j = 0; j < a.ncols(); j++)
      std::cout << a(i, j) << ",\t";
    std::cout << std::endl;
  }
  std::cout << "-----------------------" << std::endl;
}

template <typename T>
void print(const device_image2d<T>& a)
{
  if (a.nrows() * a.ncols() > 20)
    return;
  host_image2d<T> tmp(a.domain());
  copy(a, tmp);
  print(tmp);
}


int main()
{

  srand(time(0));
  obox2d<point2d<int> > domain(IMG_SIZE, IMG_SIZE);
  device_image2d<VTYPE> img(domain);
  device_image2d<VTYPE> img_conv(domain);
  host_image2d<VTYPE> img_conv_h(domain);

  host_image2d<VTYPE> imgh(domain);
  host_image2d<VTYPE> imgh_conv(domain);

  reset(imgh);

  for(unsigned i = 0; i < imgh.nrows(); i++)
    for(unsigned j = 0; j < imgh.ncols(); j++)
      for (unsigned k = 0; k < VTYPE::size; k++)
        imgh(point2d<int>(i, j))[k] = j;

  print(imgh);

//  imgh(point2d<int>(0, 0)).x = 1;
//  imgh(point2d<int>(0, 0)).x = 1;
//  imgh(point2d<int>(100, 100)).x = 1;

  copy(imgh, img);
  copy(img, img_conv_h);

  stats_diff(imgh, img_conv_h);

  print(img);
  float naive_cpu_time,
        gpu_static_global, gpu_static_texture, gpu_texture_texture, gpu_loop_texture, gpu_unroll_texture,gpu_unroll_texture_special, gpu_cuimg;

  float* weights = new float[KERNEL_SIZE];
  for (unsigned i = 0; i < KERNEL_SIZE; i++)
    weights[i] = 1.f;

  {
    clock_t t = clock();
    for (unsigned i = 0; i < 2; i++)
      convolve_cpu(imgh, imgh_conv, weights);

    naive_cpu_time = (clock() - t) / float(2* CLOCKS_PER_SEC);
    std::cout << "naive cpu convolution: " << naive_cpu_time << std::endl;
  }

  print(imgh_conv);


  float* weights_cuda;
  hipMalloc(&weights_cuda, KERNEL_SIZE * sizeof(float));
  hipMemcpy(weights_cuda, weights, KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
   check_cuda_error();
  i_int2* dpoint_cuda;
  hipMalloc(&dpoint_cuda, KERNEL_SIZE * sizeof(i_int2));
  hipMemcpy(dpoint_cuda, DPOINTS_CPU, KERNEL_SIZE * sizeof(i_int2), hipMemcpyHostToDevice);
  check_cuda_error();

  // Bind input texture.
  bindTexture2d(img, tex2d<VTYPE_CUDA>::tex());

  { // Bind dpoint texture.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
    hipBindTexture(0, kernel_dpoints, (void*)dpoint_cuda, channelDesc, KERNEL_SIZE * sizeof(i_int2));
    check_cuda_error();
  }
  { // Bind weights texture.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipBindTexture(0, kernel_weights, (void*)weights_cuda, channelDesc, KERNEL_SIZE * sizeof(float));
    check_cuda_error();
  }

  unsigned d = 16;
  dim3 dimBlock(d, d);
  dim3 dimGrid(std::ceil(img.ncols() / float(d)), std::ceil(img.nrows() / float(d)));

  {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < 20; i++)
      convolve_rows_static_global<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv), weights_cuda);
    check_cuda_error();
    hipDeviceSynchronize();
    check_cuda_error();
    gpu_static_global = (clock() - t) / float(20* CLOCKS_PER_SEC);
    std::cout << "dpoints: static, weights: global: " << gpu_static_global << std::endl;
  }
  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < 20; i++)
      convolve_rows_static_texture<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    check_cuda_error();
    hipDeviceSynchronize();
    check_cuda_error();
    gpu_static_texture = (clock() - t) / float(20* CLOCKS_PER_SEC);
    std::cout << "dpoints: static, weights: texture: " << gpu_static_texture << std::endl;
  }
  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
   for (unsigned i = 0; i < ITERATIONS_GPU; i++)
     convolve_rows_texture_texture<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv), KERNEL_SIZE);
    check_cuda_error();
    hipDeviceSynchronize();
    gpu_texture_texture = (clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "dpoints: texture, weights: texture: " << gpu_texture_texture << std::endl;
  }
  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
   for (unsigned i = 0; i < ITERATIONS_GPU; i++)
     convolve_rows_loop_texture<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    check_cuda_error();
    hipDeviceSynchronize();
    gpu_loop_texture = (clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "dpoints: loop, weights: texture: " << gpu_loop_texture << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < ITERATIONS_GPU; i++)
     convolve_rows_unrolled_texture<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    check_cuda_error();
    hipDeviceSynchronize();
    gpu_unroll_texture = (clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "dpoints: unrolled loop, weights: texture: " << gpu_unroll_texture << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < ITERATIONS_GPU; i++)
     convolve_rows_unrolled_texture_special<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    check_cuda_error();
    hipDeviceSynchronize();
    gpu_unroll_texture_special = (clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "dpoints: unrolled loop special, weights: texture: " << gpu_unroll_texture_special << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
    weigthed_window ww((point2d<int>*)DPOINTS_CPU, weights, KERNEL_SIZE);
    for (unsigned i = 0; i < ITERATIONS_GPU; i++)
     convolve(img, img_conv, ww);
    check_cuda_error();
    hipDeviceSynchronize();
    gpu_cuimg = (clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "cuimg: " << gpu_cuimg << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);


  std::cout << "Speed up: " << std::endl;
  std::cout << "dpoints: static, weights: global : " << (naive_cpu_time / gpu_static_global) << " x" << std::endl;
  std::cout << "dpoints: static, weights: texture: " << (naive_cpu_time / gpu_static_texture) << " x" << std::endl;
  std::cout << "dpoints: texture, weights: texture: " << (naive_cpu_time / gpu_texture_texture) << " x" << std::endl;
  std::cout << "dpoints: loop, weights: texture: " << (naive_cpu_time / gpu_loop_texture) << " x" << std::endl;
  std::cout << "dpoints: unroll, weights: texture: " << (naive_cpu_time / gpu_unroll_texture) << " x" << std::endl;
  std::cout << "dpoints: unroll special , weights: texture: " << (naive_cpu_time / gpu_unroll_texture_special) << " x" << std::endl;
  std::cout << "cuimg: " << (naive_cpu_time / gpu_cuimg) << " x" << std::endl;
}
