#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <cmath>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>


#include <hip/hip_runtime.h>

#include <cuimg/improved_builtin.h>
#include <cuimg/builtin_math.h>
#include <cuimg/image2d.h>
#include <cuimg/copy.h>
#include <cuimg/kernel_image2d.h>
#include <cuimg/host_image2d.h>
#include <cuimg/neighb2d_data.h>
#include <cuimg/neighb_iterator2d.h>
#include <cuimg/static_neighb2d.h>
#include <cuimg/texture.h>
#include <cuimg/convolve.h>
//#include <cuimg/meta_gaussian_coef.h>
#include <cuimg/meta_gaussian/meta_gaussian_coef_3.h>
#include <cuimg/meta_gaussian/meta_gaussian_coef_100.h>

using namespace cuimg;

#define IMG_SIZE 3072
#define KERNEL_SIZE 80
#define KERNEL_HALF_SIZE_M (KERNEL_SIZE / 2)
#define BLOCKDIM_X (32)
#define BLOCKDIM_Y (1)
#define PIX_PER_THREAD 8
#define APRON_BLOCK 1
#define VTYPE_CUDA float2
#define VTYPE i_float2
#define CONCAT(A, B, C) A ## B ## C
#define XCONCAT(A, B, C) CONCAT(A, B, C)
#define DPOINTS_CPU XCONCAT(c, KERNEL_SIZE ,_rows_cpu)
#define DPOINTS_GPU XCONCAT(c, KERNEL_SIZE ,_rows)
#define ITERATIONS_GPU 10

template <typename T>
struct tex2d;
REGISTER_TEXTURE2D_PROXY(tex2d);


__constant__ const int c3_rows[3][2] = {{-1, 0}, {0, 0}, {1, 0}};
         const int c3_rows_cpu[3][2] = {{-1, 0}, {0, 0}, {1, 0}};

//__constant__ const int c9_rows[9][2] = {{0, -4}, {0, -3}, {0, -2}, {0, -1}, {0, 0}, {0, 1}, {0, 2}, {0, 3}, {0, 4}};
//         const int c9_rows_cpu[9][2] = {{0, -4}, {0, -3}, {0, -2}, {0, -1}, {0, 0}, {0, 1}, {0, 2}, {0, 3}, {0, 4}};

__constant__ const int c9_rows[9][2] = {{-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}};
         const int c9_rows_cpu[9][2] = {{-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}};

__constant__ const int c15_rows[15][2] = {{-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}};
         const int c15_rows_cpu[15][2] = {{-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}};

__constant__ const int c31_rows[31][2] = {{-15, 0}, {-14, 0}, {-13, 0}, {-12, 0}, {-10, 0}, {-9, 0}, {-8, 0}, {-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0},{-1, 0},
                                  {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}, {8, 0}, {9, 0}, {10, 0}, {11, 0}, {12, 0}, {13, 0}, {14, 0}, {15, 0}};
         const int c31_rows_cpu[31][2] = {{-7, 0}, {-6, 0}, {-5, 0}, {-4, 0}, {-3, 0}, {-2, 0}, {-1, 0}, {0, 0}, {1, 0}, {2, 0}, {3, 0}, {4, 0}, {5, 0}, {6, 0}, {7, 0}};


texture<float, 1, hipReadModeElementType> kernel_weights;
texture<int2, 1, hipReadModeElementType> kernel_dpoints;


template <int R, int E, int N, int SIGMA>
struct gaussian_row_loop
{
  template <typename U>
  static __device__ inline U iter(const kernel_image2d<U>& out, const i_int2& p)
  {;
    return U(tex2D(tex2d<typename U::cuda_bt>::tex(), p.y + R, p.x)) * meta_gaussian_coef<N, SIGMA, R>::coef() +
    gaussian_row_loop<R + 1, E, N, SIGMA>::iter(out, p);
  }
};

template <int E, int N, int SIGMA>
struct gaussian_row_loop<E, E, N, SIGMA>
{
  template <typename U>
  static __device__ inline U iter(const kernel_image2d<U>& out, const i_int2& p)
  {;
    return U(tex2D(tex2d<typename U::cuda_bt>::tex(), p.y, p.x + E)) * meta_gaussian_coef<N, SIGMA, E>::coef();
  }
};

template <typename T, int N, int SIGMA, int KERNEL_HALF_SIZE>
__global__ void gausssian_row_static(kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);
  if (!out.has(p))
    return;
  out(p) = gaussian_row_loop<-KERNEL_HALF_SIZE, KERNEL_HALF_SIZE, N, SIGMA>::iter(out, p);
}


template <typename T, int N>
__global__ void gausssian_row_static_sm(kernel_image2d<T> in, kernel_image2d<T> out)
{
  __shared__ T s_data[BLOCKDIM_X * (PIX_PER_THREAD + APRON_BLOCK * 2)];

  int br = blockIdx.y * blockDim.y;
  int bc = blockIdx.x * PIX_PER_THREAD * blockDim.x;
/*
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);
  if (!out.has(p))
    return;
*/
  const int minx = bc - (BLOCKDIM_X * APRON_BLOCK) + threadIdx.x;

  T* in_ = (T*)((char*)in.data() + br * in.pitch() + minx * sizeof(T));
  T* out_ = (T*)((char*)out.data() + br * out.pitch() + minx * sizeof(T));

  // main data
  #pragma unroll
  for (int i = APRON_BLOCK; i < PIX_PER_THREAD + APRON_BLOCK; i++)
    s_data[threadIdx.x + i * BLOCKDIM_X] = in_[i * BLOCKDIM_X];

  // left apron
  {
  #pragma unroll
  for (int i = 0;
       i < APRON_BLOCK; i++)
     s_data[threadIdx.x + i * BLOCKDIM_X] = (minx + i * BLOCKDIM_X >= 0) ?
                in_[i * BLOCKDIM_X] : zero();
  }

  {
  // right apron
  #pragma unroll
  for (int i = APRON_BLOCK + PIX_PER_THREAD;
       i < APRON_BLOCK + APRON_BLOCK + PIX_PER_THREAD; i++)
     s_data[threadIdx.x + i * BLOCKDIM_X] =
         (minx  + i * BLOCKDIM_X < in.ncols()) ?
                in_[i * BLOCKDIM_X] : zero();
  }

// convolution
  __syncthreads();
 T res = zero();
 #pragma unroll
 for (int pc = APRON_BLOCK; pc < PIX_PER_THREAD + APRON_BLOCK; pc++)
 {
  #pragma unroll
  for (int i = 0; i < KERNEL_HALF_SIZE_M * 2 + 1; i++)
    res += s_data[pc * BLOCKDIM_X + threadIdx.x + i] * 3.f;
  out_[pc * BLOCKDIM_X] = res;
 }
}


template <typename T>
void reset(host_image2d<T>& in)
{
  memset(in.data(), 0, in.domain().nrows() * in.domain().ncols() * sizeof(T));
}

template <typename T>
void reset(device_image2d<T>& in)
{
  hipMemset(in.data(), 0, in.domain().nrows() * in.pitch());
}

template <typename T>
float diff(host_image2d<T>& a, host_image2d<T>& b)
{
  float res = 0;
  for(unsigned i = 0; i < a.nrows(); i++)
    for(unsigned j = 0; j < a.ncols(); j++)
      res += norml2(a(i, j) - b(i, j));
  return res;
}

template <typename T>
void stats_diff(host_image2d<T>& a, host_image2d<T>& b)
{
  float max = 0;
  float sum = 0;
  for(unsigned i = 0; i < a.nrows(); i++)
    for(unsigned j = 0; j < a.ncols(); j++)
    {
      float d = norml2(a(i, j) - b(i, j));
      sum += d;
      if (d > max) max = d;
    }

  std::cout << "max diff per pixel: " << max << std::endl;
  std::cout << "sum: " << sum << std::endl;
  std::cout << "mean: " << sum / (a.nrows() * a.ncols()) << std::endl;
}

template <typename T>
void print(const host_image2d<T>& a)
{
  if (a.nrows() * a.ncols() > 20)
    return;

  std::cout << "-----------------------" << std::endl;
  for(unsigned i = 0; i < a.nrows(); i++)
  {
    for(unsigned j = 0; j < a.ncols(); j++)
      std::cout << a(i, j) << ",\t";
    std::cout << std::endl;
  }
  std::cout << "-----------------------" << std::endl;
}

template <typename T>
void print(const device_image2d<T>& a)
{
  if (a.nrows() * a.ncols() > 20)
    return;
  host_image2d<T> tmp(a.domain());
  copy(a, tmp);
  print(tmp);
}


int main()
{

  srand(time(0));
  obox2d domain(IMG_SIZE, IMG_SIZE);
  device_image2d<VTYPE> img(domain);
  device_image2d<VTYPE> img_conv(domain);
  host_image2d<VTYPE> img_conv_h(domain);

  host_image2d<VTYPE> imgh(domain);
  host_image2d<VTYPE> imgh_conv(domain);

  reset(imgh);

  for(unsigned i = 0; i < imgh.nrows(); i++)
    for(unsigned j = 0; j < imgh.ncols(); j++)
      for (unsigned k = 0; k < VTYPE::size; k++)
        imgh(point2d<int>(i, j))[k] = j;

  print(imgh);

  copy(imgh, img);
  copy(img, img_conv_h);

  stats_diff(imgh, img_conv_h);

  print(img);

  float* weights = new float[KERNEL_SIZE];
  for (unsigned i = 0; i < KERNEL_SIZE; i++)
    weights[i] = 1.f;

  float* weights_cuda;
  hipMalloc(&weights_cuda, KERNEL_SIZE * sizeof(float));
  hipMemcpy(weights_cuda, weights, KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice);
   check_cuda_error();
  i_int2* dpoint_cuda;
  hipMalloc(&dpoint_cuda, KERNEL_SIZE * sizeof(i_int2));
 // hipMemcpy(dpoint_cuda, DPOINTS_CPU, KERNEL_SIZE * sizeof(i_int2), hipMemcpyHostToDevice);
  check_cuda_error();


  // Bind input texture.
  bindTexture2d(img, tex2d<VTYPE_CUDA>::tex());

  { // Bind dpoint texture.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<int2>();
    hipBindTexture(0, kernel_dpoints, (void*)dpoint_cuda, channelDesc, KERNEL_SIZE * sizeof(i_int2));
    check_cuda_error();
  }
  { // Bind weights texture.
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipBindTexture(0, kernel_weights, (void*)weights_cuda, channelDesc, KERNEL_SIZE * sizeof(float));
    check_cuda_error();
  }



  float gpu_unrolled_static;
  {
    hipDeviceSynchronize();
    dim3 dimBlock(16, 16);
    dim3 dimGrid(std::ceil(img.ncols() / float(dimBlock.x)), std::ceil(img.nrows() / float(dimBlock.y)));
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < ITERATIONS_GPU; i++)
     // gausssian_row_static<VTYPE, 0, 100, 91><<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
      gausssian_row_static<VTYPE, 0, 3, KERNEL_HALF_SIZE_M><<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    check_cuda_error();
    hipDeviceSynchronize();
    check_cuda_error();
    gpu_unrolled_static = 1000*(clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "row, unrolled, static: " << gpu_unrolled_static << std::endl;
  }

  float gpu_unrolled_static_sm;
  {
    assert( BLOCKDIM_X * APRON_BLOCK >= KERNEL_HALF_SIZE_M );
    assert( img.ncols() % (PIX_PER_THREAD * BLOCKDIM_X) == 0 );

    hipDeviceSynchronize();
    dim3 dimBlock(BLOCKDIM_X, 1);
    dim3 dimGrid(std::ceil(img.ncols() / float(dimBlock.x)) / PIX_PER_THREAD, std::ceil(img.nrows() / float(dimBlock.y)));
    reset(img_conv);
    clock_t t = clock();
//    assert(!(img.ncols() % dimBlock.x));
    for (unsigned i = 0; i < ITERATIONS_GPU; i++)
      gausssian_row_static_sm<VTYPE, 0><<<dimGrid, dimBlock>>>(mki(img), mki(img_conv));
    check_cuda_error();
    hipDeviceSynchronize();
    check_cuda_error();
    gpu_unrolled_static_sm = 1000*(clock() - t) / float(ITERATIONS_GPU* CLOCKS_PER_SEC);
    std::cout << "row, unrolled, static, shared mem: " << gpu_unrolled_static_sm << std::endl;
  }

}
