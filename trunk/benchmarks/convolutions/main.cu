#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <cmath>
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>


#include <hip/hip_runtime.h>

#include <cuimg/improved_builtin.h>
#include <cuimg/image2d.h>
#include <cuimg/image2d_math.h>
#include <cuimg/copy.h>
#include <cuimg/kernel_image2d.h>
#include <cuimg/host_image2d.h>
#include <cuimg/neighb2d_data.h>
#include <cuimg/neighb_iterator2d.h>
#include <cuimg/static_neighb2d.h>

//#include <cv.h>
//#include <highgui.h>


using namespace cuimg;

#define IMG_SIZE 3072
#define KERNEL_HALF_SIZE 1
#define KERNEL_SIZE ((KERNEL_HALF_SIZE * 2 + 1) * (KERNEL_HALF_SIZE * 2 + 1))
#define KERNEL_DPOINTS c9
#define VTYPE_CUDA float1
#define VTYPE i_float1
#define GPU_ITERATIONS 50

texture<VTYPE_CUDA, 2, hipReadModeElementType> tex;


template <typename T>
__global__ void convolve(kernel_image2d<T> in, kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!in.has(p))
    return;

  T sum = zero();
  unsigned w = 0;
  for (int i = -KERNEL_HALF_SIZE ; i <= KERNEL_HALF_SIZE ; i++)
  for (int j = -KERNEL_HALF_SIZE ; j <= KERNEL_HALF_SIZE ; j++)
  {
    i_int2 n = p + i_int2(i, j);
    if (in.has(n))
    {
      sum += in(n);
      w++;
    }
  }
  out(p) = sum / w;
}


template <typename T>
__global__ void convolve_tex(kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  T sum = zero();
  unsigned w = 0;
  for (int i = -KERNEL_HALF_SIZE ; i <= KERNEL_HALF_SIZE ; i++)
  for (int j = -KERNEL_HALF_SIZE ; j <= KERNEL_HALF_SIZE ; j++)
  {
    i_int2 n = p + i_int2(i, j);
    if (out.has(n))
    {
      sum += T(tex2D(tex, n.y, n.x));
      w++;
    }
  }
  out(p) = sum / w;
}

template <int R, int C, int E>
struct conv_c_loop
{
  template <typename U, typename T>
  static __device__ inline void iter(const kernel_image2d<U>& out, const i_int2& p, T& sum, unsigned& w)
  {
    i_int2 n = p + i_int2(R, C);
    if (out.has(n))
    {
      sum += T(tex2D(tex, n.y, n.x));
      w++;
    }
    conv_c_loop<R, C + 1, E>::iter(out, p, sum, w);
  }
};

template <int R, int E>
struct conv_c_loop<R, E, E>
{
  template <typename U, typename T>
  static __device__ inline  void iter(const kernel_image2d<U>& out, const i_int2& p, T& sum, unsigned& w)
  {
    i_int2 n = p + i_int2(R, E);
    if (out.has(n))
    {
      sum += T(tex2D(tex, float(n.y), float(n.x)));
      w++;
    }
  }
};

template <int R, int E>
struct conv_r_loop
{
  template <typename U, typename T>
  static __device__ void iter(const kernel_image2d<U>& out, const i_int2& p, T& sum, unsigned& w)
  {
    conv_c_loop<R, -KERNEL_HALF_SIZE, KERNEL_HALF_SIZE>::iter(out, p, sum, w);
    conv_r_loop<R + 1, KERNEL_HALF_SIZE>::iter(out, p, sum, w);
  }
};

template <int E>
struct conv_r_loop<E, E>
{
  template <typename U, typename T>
  static __device__ void iter(const kernel_image2d<U>& out, const i_int2& p, T& sum, unsigned& w)
  {
    conv_c_loop<E, -KERNEL_HALF_SIZE, KERNEL_HALF_SIZE>::iter(out, p, sum, w);
  }
};

template <typename T>
__global__ void convolve_tex_unrolled(kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  T sum = zero();
  unsigned w = 0;
  conv_r_loop<-KERNEL_HALF_SIZE, KERNEL_HALF_SIZE>::iter(out, p, sum, w);

  out(p) = sum / w;
}


template <typename T>
__global__ void convolve_tex_static(kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  T sum = zero();
  unsigned w = 0;
  static_neighb2d<KERNEL_SIZE> nn(KERNEL_DPOINTS);
  for (int i = 0; i < KERNEL_SIZE; i++)
  {
    i_int2 n = p + i_int2(nn[i]);
    if (out.has(n))
    {
      sum += T(tex2D(tex, n.y, n.x));
      w++;
    }
  }
  out(p) = sum / w;
}


template <typename T>
__global__ void convolve_tex_static_it(kernel_image2d<T> out)
{
  int idr = blockIdx.y * blockDim.y + threadIdx.y;
  int idc = blockIdx.x * blockDim.x + threadIdx.x;
  i_int2 p(idr, idc);

  if (!out.has(p))
    return;

  T sum = zero();
  unsigned w = 0;
  neighb_iterator2d<static_neighb2d<KERNEL_SIZE> > n(p, static_neighb2d<KERNEL_SIZE>(KERNEL_DPOINTS));
  for(n.start(); n.is_valid(); n.next() )
  {
    if (out.has(n))
    {
      sum += T(tex2D(tex, n->col(), n->row()));
      w++;
    }
  }
  out(p) = sum / w;
}



template <typename T>
void convolve_cpu(host_image2d<T>& in, host_image2d<T>& out)
{

  for (unsigned r = 0; r < in.nrows(); r++)
    for (unsigned c = 0; c < in.ncols(); c++)
    {
      point2d<int> p(r, c);

      T sum = zero();
      unsigned w = 0;
      for (int i = -KERNEL_HALF_SIZE ; i <= KERNEL_HALF_SIZE ; i++)
        for (int j = -KERNEL_HALF_SIZE ; j <= KERNEL_HALF_SIZE ; j++)
        {
          i_int2 n = i_int2(p) + i_int2(i, j);
          if (in.has(n))
          {
            sum += in(n);
            w++;
          }
        }
        out(p) = sum / w;
    }

}

template <typename T>
void reset(host_image2d<T>& in)
{
  memset(in.data(), 0, in.domain().nrows() * in.domain().ncols() * sizeof(T));
}

template <typename T>
void reset(image2d<T>& in)
{
  hipMemset(in.data(), 0, in.domain().nrows() * in.pitch());
}

template <typename T>
float diff(host_image2d<T>& a, host_image2d<T>& b)
{
  float res = 0;
  for(unsigned i = 0; i < a.nrows(); i++)
    for(unsigned j = 0; j < a.ncols(); j++)
      res += norml2(a(i, j) - b(i, j));
  return res;
}

template <typename T>
void stats_diff(host_image2d<T>& a, host_image2d<T>& b)
{
  float max = 0;
  float sum = 0;
  for(unsigned i = 0; i < a.nrows(); i++)
    for(unsigned j = 0; j < a.ncols(); j++)
    {
      float d = norml2(a(i, j) - b(i, j));
      sum += d;
      if (d > max) max = d;
    }

  std::cout << "max diff per pixel: " << max << std::endl;
  std::cout << "sum: " << sum << std::endl;
  std::cout << "mean: " << sum / (a.nrows() * a.ncols()) << std::endl;
}

template <typename T>
void print(const host_image2d<T>& a)
{
  if (a.nrows() * a.ncols() > 20)
    return;

  std::cout << "-----------------------" << std::endl;
  for(unsigned i = 0; i < a.nrows(); i++)
  {
    for(unsigned j = 0; j < a.ncols(); j++)
      std::cout << a(i, j) << ",\t";
    std::cout << std::endl;
  }
  std::cout << "-----------------------" << std::endl;
}

template <typename T>
void print(const image2d<T>& a)
{
  if (a.nrows() * a.ncols() > 20)
    return;
  host_image2d<T> tmp(a.domain());
  copy(a, tmp);
  print(tmp);
}


int main()
{
  srand(time(0));
  obox2d<point2d<int> > domain(IMG_SIZE, IMG_SIZE);
  image2d<VTYPE> img(domain);
  image2d<VTYPE> img_conv(domain);
  host_image2d<VTYPE> img_conv_h(domain);

  host_image2d<VTYPE> imgh(domain);
  host_image2d<VTYPE> imgh_conv(domain);

  reset(imgh);

  for(unsigned i = 0; i < imgh.nrows(); i++)
    for(unsigned j = 0; j < imgh.ncols(); j++)
      for (unsigned k = 0; k < VTYPE::size; k++)
        imgh(point2d<int>(i, j))[k] = i;

  print(imgh);

//  imgh(point2d<int>(0, 0)).x = 1;
//  imgh(point2d<int>(0, 0)).x = 1;
//  imgh(point2d<int>(100, 100)).x = 1;

  copy(imgh, img);
  copy(img, img_conv_h);
  
  stats_diff(imgh, img_conv_h);

  print(img);
  float naive_cpu_time,
        texture_gpu_time, texture_unroll_gpu_time, texture_static_gpu_time,
        texture_static_it_gpu_time;

  {
    clock_t t = clock();
    for (unsigned i = 0; i < 1; i++)
      convolve_cpu(imgh, imgh_conv);
  
     naive_cpu_time = (clock() - t) / float(1* CLOCKS_PER_SEC);
     std::cout << "naive cpu convolution: " << naive_cpu_time << std::endl;
  }

  print(imgh_conv);

  std::cout << imgh_conv(point2d<int>(0, 0)).x << std::endl;

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VTYPE_CUDA>();;
  hipBindTexture2D(0, tex, (void*)img.data(), channelDesc, img.ncols(), img.nrows(), img.pitch());
  check_cuda_error();

  unsigned d = 16;
  dim3 dimBlock(d, d);
  dim3 dimGrid(idivup(img.ncols(), d), idivup(img.nrows(), d));

   {
     reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < GPU_ITERATIONS; i++)
      convolve_tex_unrolled<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    hipDeviceSynchronize();
    texture_unroll_gpu_time = (clock() - t) / float(GPU_ITERATIONS* CLOCKS_PER_SEC);
    std::cout << "texture unrolled gpu convolution: " << texture_unroll_gpu_time << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < GPU_ITERATIONS; i++)
      convolve_tex_static<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    hipDeviceSynchronize();
    texture_static_gpu_time = (clock() - t) / float(GPU_ITERATIONS* CLOCKS_PER_SEC);
    std::cout << "texture static gpu convolution: " << texture_static_gpu_time << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < GPU_ITERATIONS; i++)
      convolve_tex_static_it<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    hipDeviceSynchronize();
    texture_static_it_gpu_time = (clock() - t) / float(GPU_ITERATIONS* CLOCKS_PER_SEC);
    std::cout << "texture static  it gpu convolution: " << texture_static_it_gpu_time << std::endl;
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

    {
    reset(img_conv);
    clock_t t = clock();
    for (unsigned i = 0; i < GPU_ITERATIONS; i++)
      convolve_tex<<<dimGrid, dimBlock>>>(kernel_image2d<VTYPE>(img_conv));
    hipDeviceSynchronize();
    texture_gpu_time = (clock() - t) / float(GPU_ITERATIONS* CLOCKS_PER_SEC);
    std::cout << "texture gpu convolution: " << texture_gpu_time << std::endl;
    check_cuda_error();
  }

  copy(img_conv, img_conv_h);
  print(img_conv_h);
  stats_diff(imgh_conv, img_conv_h);

  std::cout << "Speed up: " << std::endl;
  std::cout << "texture gpu convolution: " << (naive_cpu_time / texture_gpu_time) << " x" << std::endl;
  std::cout << "texture unrolled gpu convolution: " << (naive_cpu_time / texture_unroll_gpu_time) << " x" << std::endl;
  std::cout << "texture static gpu convolution: " << (naive_cpu_time / texture_static_gpu_time) << " x" << std::endl;
  std::cout << "texture static it gpu convolution: " << (naive_cpu_time / texture_static_it_gpu_time) << " x" << std::endl;

}
