#include "hip/hip_runtime.h"

#include <iostream>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <cuimg/improved_builtin.h>
#include <cuimg/error.h>

#define BUFFER_SIZE (1000*100)

#define N (1000*100)

using namespace cuimg;

__global__ void wo_cuimg(float4* buffer)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id >= BUFFER_SIZE)
    return;

  float4 f = buffer[id];
  for(int i = 0; i < N; i++)
  {
    f.x = f.x + f.x;
    f.y = f.y + f.y;
    f.z = f.z + f.z;
    f.w = f.w + f.w;
  }
  buffer[id].x = f.x + 42;
  buffer[id].y = f.y + 43;
  buffer[id].z = f.z + 44;
  buffer[id].w = f.w + 45;
}

__global__ void w_cuimg(i_float4* buffer)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  if (id >= BUFFER_SIZE)
    return;

  i_float4 f = buffer[id];
  for(int i = 0; i < N; i++)
    f = f + f;
  buffer[id] = f + i_float4(42, 43, 44, 45);
}

int main()
{
  float4* buffer_cpu = new float4[BUFFER_SIZE]();
  float4* buffer;
  hipMalloc(&buffer, BUFFER_SIZE * sizeof(float4));
  cuimg::check_cuda_error();
  assert(buffer);
  hipMemset(buffer, 0, BUFFER_SIZE * sizeof(float4));
  cuimg::check_cuda_error();

  unsigned d = 16;
  dim3 dimBlock(d);
  dim3 dimGrid(std::ceil(BUFFER_SIZE / float(d)));

  size_t time = std::clock();
  wo_cuimg<<<dimGrid, dimBlock>>>(buffer);
  cuimg::check_cuda_error();
  hipMemcpy(buffer_cpu, buffer, sizeof(float4), hipMemcpyDeviceToHost);
  cuimg::check_cuda_error();
  std::cout << "pure cuda: "<< (float(clock() - time) / float(CLOCKS_PER_SEC)) << std::endl;

  hipMemset(buffer, 0, BUFFER_SIZE * sizeof(float4));

  time = std::clock();
  w_cuimg<<<dimGrid, dimBlock>>>((i_float4*)buffer);
  cuimg::check_cuda_error();
  hipMemcpy(buffer_cpu, buffer, sizeof(float4), hipMemcpyDeviceToHost);
  cuimg::check_cuda_error();
  std::cout << "cuimg: "<< (float(clock() - time) / float(CLOCKS_PER_SEC)) << std::endl;

  std::cout << buffer_cpu[0].x << ", " << buffer_cpu[0].y << ", " << buffer_cpu[0].z << ", " << buffer_cpu[0].w << std::endl;
}
